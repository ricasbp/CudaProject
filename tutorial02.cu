#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>

using namespace std;

/* Our Kernel*/
__global__ void AddIntsCUDA(int* a, int *b){
a[0] += b[0];
}

int main(){
    
    int a = 5, b = 9;
    
    //Device pointers
    int *d_a, *d_b;

    hipMalloc(&d_a, sizeof(int)); 
    hipMalloc(&d_b, sizeof(int));

    //d_a: Destination, &a: pointer to source, its a intenger, direction: host to device
    hipMemcpy(d_a, &a, sizeof(int), hipMemcpyHostToDevice); 
    hipMemcpy(d_b, &b, sizeof(int), hipMemcpyHostToDevice); 
    
    AddIntsCUDA<<<1,1>>>(d_a, d_b);

    hipMemcpy(&a, d_a, sizeof(int), hipMemcpyDeviceToHost);

    cout << "The answer is: " <<a<<endl;

    hipFree(d_a);
    hipFree(d_b);

    return 0;
}



